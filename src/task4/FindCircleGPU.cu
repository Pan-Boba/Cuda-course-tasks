#include "hip/hip_runtime.h"
#include "FindCircle.cuh"

#define KERNEL_SIZE 3
#define IMAGE_CHANNELS 3
#define BLOCK_SIZE 16

#define SEED 30

#define BLUR_SIZE 17
#define CIRCLE_THICKNESS 3
#define N 5
#define K 100
#define THRESHOLD_RADIUS 7 // 5 for inner circle
#define THRESHOLD_COUNT 600 //100 for inner circle

using namespace fourthTask;


struct Point
{
	int x, y;
	__device__ Point() {}
	__device__ Point(int _x, int _y) : x(_x), y(_y) {}

	__device__ static double dist(const Point& first, const Point& second)
	{
		double _x = first.x - second.x;
		double _y = first.y - second.y;
		return std::sqrt(_x * _x + _y * _y);
	}

	__device__ Point& operator=(const Point& in)
	{
		x = in.x;
		y = in.y;
		return *this;
	}
};


struct Matrix
{
	int height, width;
	double** elements;

	__device__ Matrix(int _height, int _width) : height(_height), width(_width)
	{
		elements = new double* [height];
		for (int i = 0; i < height; i++)
		{
			elements[i] = new double[width];
		}
	}

	template <int _height, int _width>
	__device__ Matrix(double (&_elements)[_height][_width]) : height(_height), width(_width)
	{
		elements = new double* [height];
		for (int i = 0; i < height; i++)
		{
			elements[i] = new double[width];
			for (int j = 0; j < width; j++)
			{
				elements[i][j] = _elements[i][j];
			}
		}
	}

	__device__ static double sum(const Matrix& in)
	{
		double sum = 0;
		for (int i = 0; i < in.height; ++i)
		{
			for (int j = 0; j < in.width; ++j)
			{
				sum += in.elements[i][j];
			}
		}

		return sum;
	}

	__device__ static void subtract(Matrix& in, const double element)
	{
		for (int i = 0; i < in.height; ++i)
		{
			for (int j = 0; j < in.width; ++j)
			{
				in.elements[i][j] -= element;
			}
		}
	}

	__device__ static  double mulSum(const Matrix& first, const Matrix& second)
	{
		Matrix result(first.height, first.width);

		for (int i = 0; i < first.height; i++)
		{
			for (int j = 0; j < first.width; j++)
			{
				result.elements[i][j] = first.elements[i][j] * second.elements[i][j];
			}
		}

		return sum(result);
	}

	__device__ static double mulSum(const Matrix& first, const Matrix& second, const Matrix& third)
	{
		Matrix result(first.height, first.width);

		for (int i = 0; i < first.height; i++)
		{
			for (int j = 0; j < first.width; j++)
			{
				result.elements[i][j] = first.elements[i][j] * second.elements[i][j] * third.elements[i][j];
			}
		}

		return sum(result);
	}

	__device__ static void multiply(const Matrix& first, const Matrix& second, Matrix& result)
	{
		for (int i = 0; i < first.height; i++)
		{
			for (int j = 0; j < second.width; j++)
			{
				result.elements[i][j] = 0;

				for (int k = 0; k < second.height; k++)
				{
					result.elements[i][j] += first.elements[i][k] * second.elements[k][j];
				}
			}
		}
	}

	__device__ Matrix inverse2D()
	{
		Matrix temp(2, 2);

		double det = elements[0][0] * elements[1][1] - elements[0][1] * elements[1][0];

		temp.elements[0][0] = elements[1][1] / det;
		temp.elements[0][1] = -elements[0][1] / det;
		temp.elements[1][0] = -elements[1][0] / det;
		temp.elements[1][1] = elements[0][0] / det;

		for (int i = 0; i < 2; i++)
		{
			for (int j = 0; j < 2; j++)
			{
				elements[i][j] = temp.elements[i][j];
			}
		}

		return *this;
	}
};


__host__ unsigned int* GetCircleData(const cv::Mat& inputImage, int& numOfEdges)
{
	cv::Mat grayImage;
	cv::cvtColor(inputImage, grayImage, cv::COLOR_BGR2GRAY);

	// Blur the image for better edge detection
	cv::Mat bluredImage;
	cv::GaussianBlur(grayImage, bluredImage, cv::Size(BLUR_SIZE, BLUR_SIZE), 0);

	// https://en.wikipedia.org/wiki/Canny_edge_detector
	cv::Mat edges;
	cv::Canny(bluredImage, edges, 100, 200);

	// Save filtered image
	cv::imwrite("./src/task4/FilteredImage.png", edges);

	// Create array of edge Points
	unsigned int* edgesPoints = new unsigned int[2 * edges.rows * edges.cols];

	for (int row = 0; row < edges.rows; ++row)
	{
		for (int col = 0; col < edges.cols; ++col)
		{
			if (edges.data[row * edges.cols + col] == 255)
			{
				edgesPoints[numOfEdges] = col;
				edgesPoints[numOfEdges + 1] = row;
				numOfEdges += 2;
			}
		}
	}

	return edgesPoints;
}


__device__ Point* GetAllPoints(const unsigned int* allEdges, const int numOfAllPoints)
{
	Point* allPoints = new Point[numOfAllPoints];
	int j = 0;

	for (int i = 0; i < numOfAllPoints; i++)
	{
		
		allPoints[i] = Point(allEdges[j], allEdges[j + 1]);
		j += 2;
	}

	return allPoints;
}


__device__ void SwapRandomPoints(Point* allPoints, const int index1, const int index2)
{
	Point temp = allPoints[index1];
	allPoints[index1] = allPoints[index2];
	allPoints[index2] = temp;
}


__device__ Point* GetRandomPoints(Point* allPoints, const int numOfAllPoints)
{
	hiprandState s;
	hiprand_init(SEED, 0, 0, &s);

	for (int i = 0; i < numOfAllPoints; i++)
	{
		int randomIndex = numOfAllPoints * hiprand_uniform(&s) - 1;
		SwapRandomPoints(allPoints, i, randomIndex);
		randomIndex = numOfAllPoints * hiprand_uniform(&s) - 1;
		SwapRandomPoints(allPoints, numOfAllPoints - i, randomIndex);
	}

	Point* chosenPoints = new Point[N];

	for (int i = 0; i < N; i++)
	{
		chosenPoints[i] = allPoints[i];
	}

	return chosenPoints;
}


__device__ void GetCircleParametersLeastSquares(const Point* points, const int numOfPoints, Point& circleCenter, double& circleRadius)
{
	// https://dtcenter.org/sites/default/files/community-code/met/docs/write-ups/circle_fit.pdf
	Matrix ui(1, numOfPoints), vi(1, numOfPoints), temp(1, numOfPoints);

	for (int i = 0; i < numOfPoints; ++i)
	{
		ui.elements[0][i] = (double) points[i].x;
		vi.elements[0][i] = (double) points[i].y;
	}

	double averageX = Matrix::sum(ui) / (double)numOfPoints;
	double averageY = Matrix::sum(vi) / (double)numOfPoints;

	Matrix::subtract(ui, averageX);
	Matrix::subtract(vi, averageY);

	double Suu = Matrix::mulSum(ui, ui);
	double Svv = Matrix::mulSum(vi, vi);
	double Suv = Matrix::mulSum(ui, vi);

	double Suuu = Matrix::mulSum(ui, ui, ui);
	double Svvv = Matrix::mulSum(vi, vi, vi);
	double Suvv = Matrix::mulSum(ui, vi, vi);
	double Svuu = Matrix::mulSum(vi, ui, ui);

	double rightSideArray[2][1] = { 0.5 * (Suuu + Suvv), 0.5 * (Svvv + Svuu) };
	double leftSideArray[2][2] = { {Suu, Suv}, {Suv, Svv} };

	Matrix rightSideVector(rightSideArray);
	Matrix leftSideMatrix(leftSideArray);

	Matrix centerVector(2, 1);
	Matrix::multiply(leftSideMatrix.inverse2D(), rightSideVector, centerVector);
	
	double dataX = centerVector.elements[0][0], dataY = centerVector.elements[1][0];

	circleCenter = Point((int)(dataX + averageX), (int)(dataY + averageY));
	circleRadius = (int)std::sqrt(dataX * dataX + dataY * dataY + (Suu + Svv) / numOfPoints);
}


__device__ Point* GetAllInlinePoints(const Point* allPoints, const int numOfAllPoints, const Point circleCenter, const double circleRadius, int& numOfInlinePoints)
{
	Point* inlinePoints = new Point[numOfAllPoints];

	for (int i = 0; i < numOfAllPoints; ++i)
	{
		double pointRadius = Point::dist(allPoints[i], circleCenter);

		if ((pointRadius > (circleRadius - THRESHOLD_RADIUS)) && (pointRadius < (circleRadius + THRESHOLD_RADIUS)))
		{
			inlinePoints[numOfInlinePoints] = allPoints[i];
			++numOfInlinePoints;
		}
	}

	if (numOfInlinePoints != 0)
	{
		return inlinePoints;
	}

	return new Point();
}


__device__ double CalculateMeanError(const Point* inlinePoints, const int numOfInlinePoints, const Point circleCenter, const double circleRadius)
{
	double error = 0;

	for (int i = 0; i < numOfInlinePoints; ++i)
	{
		double pointRadius = Point::dist(inlinePoints[i], circleCenter);
		error += std::abs(circleRadius - pointRadius);
	}

	return error / numOfInlinePoints;
}


__device__ void GetCandidateCircleParameters(unsigned int* allEdges, const int numOfAllPoints, Point& circleCenter, double& circleRadius, double& error)
{
	// https://sdg002.github.io/ransac-circle/index.html
	Point* allPoints = GetAllPoints(allEdges, numOfAllPoints);
	Point* chosenEdges = GetRandomPoints(allPoints, numOfAllPoints);

	GetCircleParametersLeastSquares(chosenEdges, N, circleCenter, circleRadius);

	int numOfInlinePoints = 0;
	Point* inlinePoints = GetAllInlinePoints(allPoints, numOfAllPoints, circleCenter, circleRadius, numOfInlinePoints);

	if (numOfInlinePoints > THRESHOLD_COUNT)
	{
		GetCircleParametersLeastSquares(inlinePoints, numOfInlinePoints, circleCenter, circleRadius);

		numOfInlinePoints = 0;
		inlinePoints = GetAllInlinePoints(allPoints, numOfAllPoints, circleCenter, circleRadius, numOfInlinePoints);

		if (numOfInlinePoints > THRESHOLD_COUNT)
		{
			error = CalculateMeanError(inlinePoints, numOfInlinePoints, circleCenter, circleRadius);
			return;
		}
	}

	error = -1;
}


__global__ void IterateOverCandidates(unsigned int* edgesPoints, const int numOfAllPoints, Point * circleCenters, double* circleRadiuses, double* errors, int& numOfCandidates)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < K)
	{
		Point circleCenter;
		double circleRadius, error;
		GetCandidateCircleParameters(edgesPoints, numOfAllPoints, circleCenter, circleRadius, error);

		if (error != -1)
		{
			circleCenters[numOfCandidates] = circleCenter;
			circleRadiuses[numOfCandidates] = circleRadius;
			errors[numOfCandidates] = error;
			++numOfCandidates;
		}
	}
}


__host__ std::pair<cv::Point, double> ChooseBestParameters(const Point* circleCenters, const double* circleRadiuses, const double* errors, const int numOfCandidates)
{
	assert(numOfCandidates != 0);

	int bestParameterIndex = 0;
	double tempError = 10000;

	for (int i = 0; i < numOfCandidates; ++i)
	{
		if (errors[i] < tempError)
		{
			tempError = errors[i];
			bestParameterIndex = i;
		}
	}

	std::pair <cv::Point, double> circleParameters(cv::Point(circleCenters[bestParameterIndex].x, circleCenters[bestParameterIndex].y), circleRadiuses[bestParameterIndex]);

	return circleParameters;
}


__host__ std::pair<cv::Point, double> GetOptimalParameters(unsigned int* edgesPoints, const int numOfEdges)
{
	int numOfCandidates = 0;
	Point* circleCenters, * circleCentersCPU;
	double* circleRadiuses, * circleRadiusesCPU = new double[K], * errors, * errorsCPU = new double[K];

	hipMallocManaged(&circleCenters, K * sizeof(Point));
	hipMallocManaged(&circleRadiuses, K * sizeof(double));
	hipMallocManaged(&errors, K * sizeof(double));

	dim3 blockSize(BLOCK_SIZE);
	dim3 numBlocks(ceil(K / (double) BLOCK_SIZE));

	IterateOverCandidates <<<numBlocks, blockSize>>> (edgesPoints, (int)(numOfEdges / 2), circleCenters, circleRadiuses, errors, numOfCandidates);
	hipDeviceSynchronize();

	hipMemcpy(errors, errorsCPU, K, hipMemcpyDeviceToHost);
	hipMemcpy(circleCenters, circleCentersCPU, K, hipMemcpyDeviceToHost);
	hipMemcpy(circleRadiuses, circleRadiusesCPU, K, hipMemcpyDeviceToHost);

	// Wait for GPU to finish
	hipDeviceSynchronize();

	return ChooseBestParameters(circleCentersCPU, circleRadiusesCPU, errorsCPU, numOfCandidates);
}


void fourthTask::FindCircleGPU(const std::string pathToImage)
{
	// read data
	cv::Mat inputImage = cv::imread(pathToImage);
	cv::Mat outputImage = inputImage.clone();
	int numOfEdges = 0;

	// select data
	unsigned int* edgesPoints = GetCircleData(inputImage, numOfEdges);
	unsigned int* edgesPointsGPU;

	// allocate data
	{
		hipMallocManaged(&edgesPointsGPU, numOfEdges * sizeof(unsigned int));
		hipMemcpy(edgesPoints, edgesPointsGPU, numOfEdges * sizeof(unsigned int), hipMemcpyHostToDevice);
	}

	// get optimal circle parameters
	std::pair<cv::Point, double> circleParameters = GetOptimalParameters(edgesPointsGPU, numOfEdges);

	// add circle to the image
	{
		cv::circle(outputImage, circleParameters.first, circleParameters.second, cv::Scalar(0, 0, 0), CIRCLE_THICKNESS);
		cv::imwrite("./src/task4/OutputImage.png", outputImage);
	}

	// free memorey
	{
		delete[] edgesPoints;
		hipFree(edgesPointsGPU);
	}
}
